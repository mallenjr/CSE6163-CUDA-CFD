
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

void computepi(int N, double *result) {
  double sum = 0 ;
  double dx = 1./double(N) ;
  for(int i=0;i<N;++i) {
    double x = dx*(i+0.5) ;
    double f = sqrt(1.-x*x) ;
    sum += dx*f ;
  }
  *result = 4.*sum ;
}

const int NBLOCKS = 64 ;
const int NTHREADS = 1024 ;
__global__
void pikernel(int N, double *result) {
  double sum = 0 ;
  double dx = 1./double(N) ;
  int t = threadIdx.x ;
  int nseg = N/(NBLOCKS*NTHREADS) ;
  int b = blockIdx.x ;
  for(int i=t*nseg;i<(t+1)*nseg;++i) {
    double x = dx*(i+b*nseg*NTHREADS+0.5) ;
    double f = sqrt(1.-x*x) ;
    sum += dx*f ;
  }
  __shared__ double scratch[1024] ;
  scratch[t] = 4*sum ;
  int offset = NTHREADS ;
  while(offset > 2) {
    offset = offset/2 ;
    __syncthreads() ;
    if(t < offset)
      scratch[t] += scratch[t+offset] ;
  }
  __syncthreads() ;
  if(t==0)
    result[b] = scratch[0]+scratch[1] ;
}

__global__ 
void pisum(int N, double *result) {
  __shared__ double scratch[1024] ;
  int t = threadIdx.x ;
  scratch[t] = result[t] ;
  int offset = N ;
  while(offset > 2) {
    offset = offset/2 ;
    __syncthreads() ;
    if(t < offset)
      scratch[t] += scratch[t+offset] ;
  }
  __syncthreads() ;
  if(t==0)
    result[0] = scratch[0]+scratch[1] ;
}

int main(void) {
  using namespace std ;
  int N = 1<<20 ;
  cout.precision(16) ;
  hipEvent_t start,stop ;
  hipEventCreate(&start) ;
  hipEventCreate(&stop) ;

  double pi = 0 ;
  double *d_pi ;
  hipError_t err = hipMalloc(&d_pi,NBLOCKS*sizeof(double)) ;

  if(err!= hipSuccess) {
    cerr << "hipMalloc for d_pi failed" << endl ;
    cerr << "errror = " << hipGetErrorString(err) << endl;
  }


  hipEventRecord(start) ;
  // execute kernel with 1 thread block, 1 thread per block
  pikernel<<<NBLOCKS,NTHREADS>>>(N,d_pi) ;
  err = hipGetLastError() ;
  if(err != hipSuccess) {
    cerr << "kernal launch failed: " << hipGetErrorString(err) << endl ;
  }
  pisum<<<1,NBLOCKS>>>(NBLOCKS,d_pi) ;
  hipEventRecord(stop) ;
  //  double cudapi[NTHREADS] ;
  double *cudapi = (double *)malloc(sizeof(double)*1) ;
  err = hipMemcpy(&cudapi[0],d_pi,1*sizeof(double),hipMemcpyDeviceToHost) ;
  if(err!= hipSuccess) {
    cerr << "hipMemcpy for d_pi failed" << endl ;
    cerr << "error = " << hipGetErrorString(err) << endl;
  }
  hipEventSynchronize(stop) ;
  float milliseconds = 0 ;
  hipEventElapsedTime(&milliseconds,start,stop) ;
  cout<< "pi kernel execution time = " << milliseconds << " ms" << endl ;

  double cudapisum = 0 ;
  for(int i=0;i<1;++i)
    cudapisum += cudapi[i] ;
  computepi(N,&pi) ;
  cout << "pi = " << pi << " cudapi= " << cudapisum << endl ;
}

