
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

void computepi(int N, double *result) {
  double sum = 0 ;
  double dx = 1./double(N) ;
  for(int i=0;i<N;++i) {
    double x = dx*(i+0.5) ;
    double f = sqrt(1.-x*x) ;
    sum += dx*f ;
  }
  *result = 4.*sum ;
}

const int NBLOCKS = 1 ;
const int NTHREADS = 256 ;
__global__
void pikernel(int N, double *result) {
  double sum = 0 ;
  double dx = 1./double(N) ;
  int t = threadIdx.x ;
  int nseg = N/(NBLOCKS*NTHREADS) ;
  for(int i=t*nseg;i<(t+1)*nseg;++i) {
    double x = dx*(i+0.5) ;
    double f = sqrt(1.-x*x) ;
    sum += dx*f ;
  }
  //  __shared__ double scratch[NTHREADS] ;
  result[t] = 4*sum ;
  int offset = 512 ;
  while(offset > 1) {
    __syncthreads() ;
    if(t+offset < 1024)
      result[t] += result[t+offset] ;
    offset = offset / 2 ;
  }
  __syncthreads() ;
  if(t==0)
    *result = result[0]+result[1] ;
}

int main(void) {
  using namespace std ;
  int N = 1<<20 ;
  cout.precision(16) ;
  hipEvent_t start,stop ;
  hipEventCreate(&start) ;
  hipEventCreate(&stop) ;

  double pi = 0 ;
  double *d_pi ;
  hipError_t err = hipMalloc(&d_pi,NTHREADS*sizeof(double)) ;

  if(err!= hipSuccess) {
    cerr << "hipMalloc for d_pi failed" << endl ;
    cerr << "errror = " << hipGetErrorString(err) << endl;
  }


  hipEventRecord(start) ;
  // execute kernel with 1 thread block, 1 thread per block
  pikernel<<<NBLOCKS,NTHREADS>>>(N,d_pi) ;
  err = hipGetLastError() ;
  if(err != hipSuccess) {
    cerr << "kernal launch failed: " << hipGetErrorString(err) << endl ;
  }
  hipEventRecord(stop) ;
  double cudapi[1] ;
  err = hipMemcpy(&cudapi[0],d_pi,sizeof(double),hipMemcpyDeviceToHost) ;
  if(err!= hipSuccess) {
    cerr << "hipMemcpy for d_pi failed" << endl ;
    cerr << "error = " << hipGetErrorString(err) << endl;
  }
  hipEventSynchronize(stop) ;
  float milliseconds = 0 ;
  hipEventElapsedTime(&milliseconds,start,stop) ;
  cout<< "pi kernel execution time = " << milliseconds << " ms" << endl ;

  double cudapisum = cudapi[0];
  computepi(N,&pi) ;
  cout << "pi = " << pi << " cudapi= " << cudapisum << endl ;
}

